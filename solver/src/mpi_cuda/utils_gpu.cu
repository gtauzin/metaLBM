extern "C" {
#include "utils_gpu.h"
}

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

namespace lbm {

  extern "C"
  void Init_DeviceLattice(double *f, double *d_f, const int size) {

    cudaErrorCheck(hipMalloc((void **) &d_f, size));
    cudaErrorCheck(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
  }

  extern "C"
  void Free_DeviceLattice(double *d_f){
    cudaErrorCheck(hipFree(d_f));
  }

  extern "C"
  void Copy_LatticeFromDevice(double *d_f, double *f, const int size) {
    cudaErrorCheck(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));
  }

  extern "C"
  void Copy_LatticeToDevice(double *d_f, double *f, const int size) {
    cudaErrorCheck(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
  }

}
